#include "hip/hip_runtime.h"
#include <stdio.h>

// Huffman Node Structure
struct Node {
    unsigned char data;
    unsigned frequency;
    Node *left, *right;
};

// Kernel function to calculate frequencies
__global__ void calculateFrequencies(const unsigned char* data, unsigned int* frequencies, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
        atomicAdd(&frequencies[data[idx]], 1);
}

// Helper function to create a new node
Node* createNode(unsigned char data, unsigned frequency, Node* left, Node* right) {
    Node* node = new Node;
    node->data = data;
    node->frequency = frequency;
    node->left = left;
    node->right = right;
    return node;
}

// Comparison function for sorting nodes based on frequency
int compareNodes(const void* a, const void* b) {
    Node** nodeA = (Node**)a;
    Node** nodeB = (Node**)b;
    return (*nodeA)->frequency - (*nodeB)->frequency;
}

// Build Huffman tree and generate codes
void buildHuffmanTree(Node** nodes, int size) {
    while (size > 1) {
        Node* left = nodes[0];
        Node* right = nodes[1];
        Node* parent = createNode(0, left->frequency + right->frequency, left, right);

        nodes[0] = parent;
        for (int i = 1; i < size - 1; ++i)
            nodes[i] = nodes[i + 1];
        nodes[size - 1] = nullptr;
        --size;

        qsort(nodes, size, sizeof(Node*), compareNodes);
    }
}

// Recursive function to generate Huffman codes
void generateHuffmanCodes(Node* root, unsigned int code, unsigned int codeLength, unsigned int* codes) {
    if (root->left) {
        generateHuffmanCodes(root->left, code << 1, codeLength + 1, codes);
    }
    if (root->right) {
        generateHuffmanCodes(root->right, (code << 1) | 1, codeLength + 1, codes);
    }
    if (!root->left && !root->right) {
        codes[root->data] = code << (32 - codeLength);
    }
}

// Huffman encoding function
void huffmanEncode(const unsigned char* data, int size, unsigned int* codes) {
    const int alphabetSize = 256;

    // Initialize frequencies to 0
    unsigned int* frequencies;
    hipMallocManaged(&frequencies, alphabetSize * sizeof(unsigned int));
    hipMemset(frequencies, 0, alphabetSize * sizeof(unsigned int));

    // Calculate frequencies
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    calculateFrequencies<<<blocksPerGrid, threadsPerBlock>>>(data, frequencies, size);
    hipDeviceSynchronize();

    // Create nodes for each symbol with non-zero frequency
    Node** nodes = new Node*[alphabetSize];
    int nodeCount = 0;
    for (int i = 0; i < alphabetSize; ++i) {
        if (frequencies[i] > 0) {
            nodes[nodeCount++] = createNode(i, frequencies[i], nullptr, nullptr);
        }
    }

    // Sort nodes based on frequency
    qsort(nodes, nodeCount, sizeof(Node*), compareNodes);

    // Build Huffman tree
    buildHuffman
